#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>
#include <stdio.h>


using namespace std;
texture<float, 1, hipReadModeElementType> texreference;

__global__ void kernel(float* doarray, int size)
{

    //calculate each thread global index
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    //int y = blockIdx.y*blockDim.y + threadIdx.y;

    //int offset = x + y*blockDim.x*gridDim.x;
    //fetch global memory through texture reference
    doarray[x] = tex1Dfetch(texreference, x);
    return;
}



int main(int argc, char** argv)
{
    int size = 64;
    float* harray;
    float* oarray;
    float* diarray;
    float* doarray;
    //allocate host and device memory
    harray = (float*)malloc(sizeof(float)*size);
    oarray = (float*)malloc(sizeof(float)*size);
    hipMalloc((void**)&diarray, sizeof(float)*size);
    hipMalloc((void**)&doarray, sizeof(float)*size);
    //initialize host array before usage
    for (int loop = 0; loop<size; loop++)
        harray[loop] = (float)loop;
    //copy array from host to device memory
    hipMemcpy(diarray, harray, sizeof(float)*size, hipMemcpyHostToDevice);
    //bind texture reference with linear memory
    hipBindTexture(0, texreference, diarray, sizeof(float)*size);
    //execute device kernel
    kernel << <(int)ceil((float)size / 64), 64 >> >(doarray, size);
    //unbind texture reference to free resource
    hipUnbindTexture(texreference);
    //copy result array from device to host memory
    hipMemcpy(oarray, doarray, sizeof(float)*size, hipMemcpyDeviceToHost);
    //free host and device memory
    for(int i = 0; i < size; i++){
      printf("%f\n", oarray[i]);
    }
    free(harray);
    free(oarray);
    hipUnbindTexture(&texreference);
    hipFree(diarray);
    hipFree(doarray);
    return 0;
}
